
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <unistd.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <iostream>
#include "omp.h"

//See values of N in assignment instructions.
#define N 100000
//Do not change the seed, or your answer will not be correct
#define SEED 72

//For GPU implementation
#define BLOCKSIZE 1024


struct pointData{
double x;
double y;
};



void generateDataset(struct pointData * data);

using namespace std;
void warmUpGPU();
__global__ void epsilonDist(struct pointData * data, double epsilon, int * numPairsInE);


int main(int argc, char *argv[])
{	
	//WARM UP///////////
	warmUpGPU();
	
	//Read epsilon distance from command line
	if (argc!=2)
	{
	printf("\nIncorrect number of input parameters. Please input an epsilon distance.\n");
	return 0;
	}

	
	char inputEpsilon[20];
	strcpy(inputEpsilon,argv[1]);
	double epsilon=atof(inputEpsilon);
	
	

	//generate dataset:
	struct pointData * data;
	data=(struct pointData*)malloc(sizeof(struct pointData)*N);
	printf("\nSize of dataset (MiB): %f",(2.0*sizeof(double)*N*1.0)/(1024.0*1024.0));
	generateDataset(data);
	

	//Write your code here:
	//The data you need to use is stored in the variable "data", 
	//which is of type pointData
	
//Data Copy and Start of Total GPU time/////
	double tstart=omp_get_wtime();

	hipError_t errCode = hipSuccess;
	struct pointData* dev_Data;
	int * numPairsInE = (int*)malloc(sizeof(int));
	*numPairsInE = 0;
	int * dev_pairs;
	
	
    errCode=hipMalloc((struct pointData**)&dev_Data, sizeof(struct pointData)*N);	
	if(errCode != hipSuccess) {
	cout << "\nError: A error with code " << errCode << endl; 
	}
	
	errCode=hipMalloc((int**)&dev_pairs, sizeof(int));	
	if(errCode != hipSuccess) {
	cout << "\nError: A error with code " << errCode << endl; 
	}


	errCode=hipMemcpy( dev_Data, data, sizeof(struct pointData)*N, hipMemcpyHostToDevice);
	if(errCode != hipSuccess) {
	cout << "\nError: A memcpy error with code " << errCode << endl; 
	}	
	
	errCode=hipMemcpy( dev_pairs, numPairsInE, sizeof(int), hipMemcpyHostToDevice);
	if(errCode != hipSuccess) {
	cout << "\nError: A memcpy error with code " << errCode << endl; 
	}	



//Data Copy End/////
	
	//Run Kernel
    const unsigned int totalBlocks=ceil(N*1.0/BLOCKSIZE);
	printf("\ntotal blocks: %d",totalBlocks);

	double tstartKernel =omp_get_wtime();
	epsilonDist<<<totalBlocks,BLOCKSIZE>>>(dev_Data, epsilon, dev_pairs);
	hipDeviceSynchronize();
	double tendKernel =omp_get_wtime();

	if(errCode != hipSuccess){
		cout<<"Error after kernel launch "<<errCode<<endl;
	}
	
	
	//Data copy 2
	hipMemcpy( numPairsInE, dev_pairs, sizeof(int), hipMemcpyDeviceToHost);
	
	hipDeviceSynchronize();
	
	printf("\nNumber of Pairs: %d", *numPairsInE);
	
	double tend=omp_get_wtime();
	printf("\nKernel time (s): %.15f",tendKernel-tstartKernel);	
	printf("\nTotal time (s): %.15f",tend-tstart);


	free(data);
	printf("\n");
	return 0;
}

__global__ void epsilonDist(struct pointData * data, double epsilon, int * numPairsInE) 
{
	
	unsigned int tid=threadIdx.x + (blockIdx.x*blockDim.x);
	if(tid>=N)
	{
		return;
	}
	
	unsigned int index1 = tid;
	unsigned int index2;
	
	for(index2 = 0; index2 < N; index2++)
	{
		if( sqrt( pow( (data[index1].x - data[index2].x), 2 )
	        + pow( (data[index1].y - data[index2].y), 2 ) 
	        )
	        <= epsilon )
		{
			atomicAdd(numPairsInE, 1);
		}
	}
	
	return;
}





__global__ void warmup(unsigned int * tmp) {
if (threadIdx.x==0)
*tmp=555;

return;
}



void warmUpGPU(){


printf("\nWarming up GPU for time trialing...\n");	
unsigned int * dev_tmp;
unsigned int * tmp;
tmp=(unsigned int*)malloc(sizeof(unsigned int));
*tmp=0;
hipError_t errCode=hipSuccess;
errCode=hipMalloc((unsigned int**)&dev_tmp, sizeof(unsigned int));	
	if(errCode != hipSuccess) {
	cout << "\nError: dev_tmp error with code " << errCode << endl; 
	}

warmup<<<1,256>>>(dev_tmp);

//copy data from device to host 
	errCode=hipMemcpy( tmp, dev_tmp, sizeof(unsigned int), hipMemcpyDeviceToHost);
	if(errCode != hipSuccess) {
	cout << "\nError: getting tmp result form GPU error with code " << errCode << endl; 
	}

	hipDeviceSynchronize();

	printf("\ntmp (changed to 555 on GPU): %d",*tmp);

hipFree(dev_tmp);

return;
}



//Do not modify the dataset generator or you will get the wrong answer
void generateDataset(struct pointData * data)
{

	//seed RNG
	srand(SEED);


	for (unsigned int i=0; i<N; i++){
		data[i].x=1000.0*((double)(rand()) / RAND_MAX);	
		data[i].y=1000.0*((double)(rand()) / RAND_MAX);	
	}
	

}
