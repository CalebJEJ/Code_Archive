
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <unistd.h>
#include <stdio.h>
#include <string.h>
#include <iostream>
#include <omp.h>

#define N 500000000

using namespace std;

void warmUpGPU();
__global__ void vectorAdd(unsigned int * A, unsigned int * B, unsigned int * C);

int main(int argc, char *argv[])
{
	
	warmUpGPU();
	
	unsigned int * A;
	unsigned int * B;
	unsigned int * C;
	unsigned int * C_CPU;

	A=(unsigned int *)malloc(sizeof(unsigned int)*N);
	B=(unsigned int *)malloc(sizeof(unsigned int)*N);
	C=(unsigned int *)malloc(sizeof(unsigned int)*N);
	C_CPU=(unsigned int *)malloc(sizeof(unsigned int)*N);


	printf("\nSize of A+B+C (GiB): %f",(sizeof(unsigned int)*N*3.0)/(1024.0*1024.0*1024.0));
	

	//init:
	int i=0;
	for (i=0; i<N; i++){
		A[i]=i;
		B[i]=i;
		C[i]=0;
		C_CPU[i]=0;
	}
    
    //Start Time (after warmup)
    double tstart=omp_get_wtime();

	//CPU version:
	
	
	for (int i=0; i<N; i++){
		C_CPU[i]=A[i]+B[i];
	}
	
    //End Time / Synchronize
    double tend=omp_get_wtime();
	printf("\nTotal time CPU (s): %f",tend-tstart);
	
	//CUDA error code:
	hipError_t errCode=hipSuccess;

	unsigned int * dev_A;
	unsigned int * dev_B;
	unsigned int * dev_C;

//GPU///////
    //Start Time (after warmup)
    tstart=omp_get_wtime();

    //Start Time (Data transfer)
    double tstartData1=omp_get_wtime();

	//allocate on the device: A, B, C
	errCode=hipMalloc((unsigned int**)&dev_A, sizeof(unsigned int)*N);	
	if(errCode != hipSuccess) {
	cout << "\nError: A error with code " << errCode << endl; 
	}

	errCode=hipMalloc((unsigned int**)&dev_B, sizeof(unsigned int)*N);	
	if(errCode != hipSuccess) {
	cout << "\nError: B error with code " << errCode << endl; 
	}

	errCode=hipMalloc((unsigned int**)&dev_C, sizeof(unsigned int)*N);	
	if(errCode != hipSuccess) {
	cout << "\nError: C error with code " << errCode << endl; 
	}

	//copy A to device
	errCode=hipMemcpy( dev_A, A, sizeof(unsigned int)*N, hipMemcpyHostToDevice);
	if(errCode != hipSuccess) {
	cout << "\nError: A memcpy error with code " << errCode << endl; 
	}	
	
	//copy B to device
	errCode=hipMemcpy( dev_B, B, sizeof(unsigned int)*N, hipMemcpyHostToDevice);
	if(errCode != hipSuccess) {
	cout << "\nError: A memcpy error with code " << errCode << endl; 
	}

	//copy C to device (initialized to 0)
	errCode=hipMemcpy( dev_C, C, sizeof(unsigned int)*N, hipMemcpyHostToDevice);
	if(errCode != hipSuccess) {
	cout << "\nError: A memcpy error with code " << errCode << endl; 
	}
    
	//End Time / Synchronize
    double tendData1=omp_get_wtime();
	double dataFromTime = tendData1-tstartData1;
	
	
	
	//execute kernel
	const unsigned int totalBlocks=ceil(N*1.0/1024.0);
	printf("\ntotal blocks: %d",totalBlocks);
	
	//Start Time (GPU exe)
    double tstartGPUKernal =omp_get_wtime();
	vectorAdd<<<totalBlocks,1024>>>(dev_A, dev_B, dev_C);
	
    //End Time / Synchronize GPU Kernel
	hipDeviceSynchronize();
    double tendGPUKernal =omp_get_wtime();
	printf("\nTotal time GPU Kernal (s): %f",tendGPUKernal-tstartGPUKernal);

	if(errCode != hipSuccess){
		cout<<"Error after kernel launch "<<errCode<<endl;
	}

    //Start Time (Data transfer 2)
    double tstartData2 = omp_get_wtime();

	//copy data from device to host 
	errCode=hipMemcpy( C, dev_C, sizeof(unsigned int)*N, hipMemcpyDeviceToHost);
	if(errCode != hipSuccess) {
	cout << "\nError: getting C result form GPU error with code " << errCode << endl; 
	}

    //End Time / Synchronize
    double tendData2 =omp_get_wtime();
	printf("\nTotal time Data Transfer (s): %f", dataFromTime + tendData2-tstartData2);

//GPU END////
	//End Time / Synchronize
    tend=omp_get_wtime();
	printf("\nTotal time GPU (s): %f",tend-tstart);

	return 0;
}

__global__ void vectorAdd(unsigned int * A, unsigned int * B, unsigned int * C) {

unsigned int tid=threadIdx.x+ (blockIdx.x*blockDim.x); 

if (tid>=N){
	return;
}
C[tid]=A[tid]+B[tid];

return;
}



__global__ void warmup(unsigned int * tmp) {
if (threadIdx.x==0)
*tmp=555;

return;
}



void warmUpGPU(){


printf("\nWarming up GPU for time trialing...\n");	
unsigned int * dev_tmp;
unsigned int * tmp;
tmp=(unsigned int*)malloc(sizeof(unsigned int));
*tmp=0;
hipError_t errCode=hipSuccess;
errCode=hipMalloc((unsigned int**)&dev_tmp, sizeof(unsigned int));	
	if(errCode != hipSuccess) {
	cout << "\nError: dev_tmp error with code " << errCode << endl; 
	}

warmup<<<1,256>>>(dev_tmp);

//copy data from device to host 
	errCode=hipMemcpy( tmp, dev_tmp, sizeof(unsigned int), hipMemcpyDeviceToHost);
	if(errCode != hipSuccess) {
	cout << "\nError: getting tmp result form GPU error with code " << errCode << endl; 
	}

	hipDeviceSynchronize();

	printf("\ntmp (changed to 555 on GPU): %d",*tmp);

hipFree(dev_tmp);

return;
}