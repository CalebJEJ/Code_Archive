
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <unistd.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <iostream>
#include "omp.h"

//See values of N in assignment instructions.
#define N 100000
//Do not change the seed, or your answer will not be correct
#define SEED 72

//For GPU implementation
#define BLOCKSIZE 1024


struct pointData{
double x;
double y;
};



void generateDataset(struct pointData * data);

using namespace std;
void warmUpGPU();
__global__ void epsilonDist(struct pointData * data, double epsilon, int * numPairsInE);


int main(int argc, char *argv[])
{	
	//WARM UP///////////
	warmUpGPU();
	
	//Read epsilon distance from command line
	if (argc!=2)
	{
	printf("\nIncorrect number of input parameters. Please input an epsilon distance.\n");
	return 0;
	}

	
	char inputEpsilon[20];
	strcpy(inputEpsilon,argv[1]);
	double epsilon=atof(inputEpsilon);
	
	

	//generate dataset:
	struct pointData * data;
	data=(struct pointData*)malloc(sizeof(struct pointData)*N);
	printf("\nSize of dataset (MiB): %f",(2.0*sizeof(double)*N*1.0)/(1024.0*1024.0));
	generateDataset(data);
	

	//Write your code here:
	//The data you need to use is stored in the variable "data", 
	//which is of type pointData
	
//Data Copy Start of Total GPU Time/////
	double tstart=omp_get_wtime();

	hipError_t errCode = hipSuccess;
	struct pointData* dev_Data;
	int * numPairsInE = (int*)malloc(sizeof(int)*N);
	for(int i = 0; i<N; i++)
	{
		numPairsInE[i] = 0;
	}
	int * dev_pairs;
	
	
    errCode=hipMalloc((struct pointData**)&dev_Data, sizeof(struct pointData)*N);	
	if(errCode != hipSuccess) {
	cout << "\nError: A error with code " << errCode << endl; 
	}
	
	errCode=hipMalloc((int**)&dev_pairs, sizeof(int)*N);	
	if(errCode != hipSuccess) {
	cout << "\nError: A error with code " << errCode << endl; 
	}


	errCode=hipMemcpy( dev_Data, data, sizeof(struct pointData)*N, hipMemcpyHostToDevice);
	if(errCode != hipSuccess) {
	cout << "\nError: A memcpy error with code " << errCode << endl; 
	}	
	
	errCode=hipMemcpy( dev_pairs, numPairsInE, sizeof(int)*N, hipMemcpyHostToDevice);
	if(errCode != hipSuccess) {
	cout << "\nError: A memcpy error with code " << errCode << endl; 
	}	



//Data Copy End/////

	//Run Kernel
    const unsigned int totalBlocks=ceil(N*1.0/BLOCKSIZE);
	printf("\ntotal blocks: %d",totalBlocks);

	double tstartKernel =omp_get_wtime();
	epsilonDist<<<totalBlocks,BLOCKSIZE>>>(dev_Data, epsilon, dev_pairs);
	hipDeviceSynchronize();
	
	//Need to transfer data back for processing, this shouldn't be added to kernel time
	double tstartData2 =omp_get_wtime();
	//Data copy 2
	hipMemcpy( numPairsInE, dev_pairs, sizeof(int)*N, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	double tendData2 =omp_get_wtime();
	//End of data transfer 2 -> resume calculation
	
	int totalPairs = 0;
	for(int i = 0; i < N; i++)
	{
		totalPairs = totalPairs + numPairsInE[i];
	}
	totalPairs = totalPairs * 2;
	totalPairs = totalPairs + N;
	
	double tendKernel =omp_get_wtime();

	if(errCode != hipSuccess){
		cout<<"Error after kernel launch "<<errCode<<endl;
	}

	
	printf("\nNumber of Pairs: %d", totalPairs);
	
	double tend=omp_get_wtime();
	                                            //Remove time used to move data back from kernel time
	printf("\nKernel time (s): %.15f",tendKernel-tstartKernel - (tstartData2 - tendData2));	
	printf("\nTotal time (s): %.15f",tend-tstart);



	free(data);
	printf("\n");
	return 0;
}

__global__ void epsilonDist(struct pointData * data, double epsilon, int * numPairsInE) 
{
	
	unsigned int tid=threadIdx.x + (blockIdx.x*blockDim.x);
	if(tid>=N)
	{
		return;
	}
	
	unsigned int index2;
	double diffX = 0;
    double diffY = 0;
	int count = 0;
	
	__shared__ struct pointData dataShared[BLOCKSIZE];

	dataShared[threadIdx.x] = data[tid];
	
	
	for(index2 = tid + 1; index2 < N; index2++)
	{
		diffX = fabs(dataShared[threadIdx.x].x - data[index2].x);
		diffY = fabs(dataShared[threadIdx.x].y - data[index2].y);
		
		//if the points are clearly too far then don't count them
		if( diffX <= epsilon && diffY <= epsilon )
		{
			if( sqrt( pow( diffX, 2 ) + pow( diffY, 2 ) )
	                    <= epsilon )
			{
				count++;
			}
		}
		
	}
	
	numPairsInE[threadIdx.x] = numPairsInE[threadIdx.x] + count;
	
	return;
}





__global__ void warmup(unsigned int * tmp) {
if (threadIdx.x==0)
*tmp=555;

return;
}



void warmUpGPU(){


printf("\nWarming up GPU for time trialing...\n");	
unsigned int * dev_tmp;
unsigned int * tmp;
tmp=(unsigned int*)malloc(sizeof(unsigned int));
*tmp=0;
hipError_t errCode=hipSuccess;
errCode=hipMalloc((unsigned int**)&dev_tmp, sizeof(unsigned int));	
	if(errCode != hipSuccess) {
	cout << "\nError: dev_tmp error with code " << errCode << endl; 
	}

warmup<<<1,256>>>(dev_tmp);

//copy data from device to host 
	errCode=hipMemcpy( tmp, dev_tmp, sizeof(unsigned int), hipMemcpyDeviceToHost);
	if(errCode != hipSuccess) {
	cout << "\nError: getting tmp result form GPU error with code " << errCode << endl; 
	}

	hipDeviceSynchronize();

	printf("\ntmp (changed to 555 on GPU): %d",*tmp);

hipFree(dev_tmp);

return;
}



//Do not modify the dataset generator or you will get the wrong answer
void generateDataset(struct pointData * data)
{

	//seed RNG
	srand(SEED);


	for (unsigned int i=0; i<N; i++){
		data[i].x=1000.0*((double)(rand()) / RAND_MAX);	
		data[i].y=1000.0*((double)(rand()) / RAND_MAX);	
	}
	

}
